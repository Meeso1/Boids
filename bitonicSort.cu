#include "hip/hip_runtime.h"
/*
 * Source: https://gist.github.com/mre/1392067
 *
 * Parallel bitonic sort using CUDA.
 * Based on http://www.tools-of-computing.com/tc/CS/Sorts/bitonic_sort.htm
 * License: BSD 3
 */

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "cudaUtils.cu"
#include "utils.cu"

#define T(s) POINT(4, s)

///
/// FLOAT
///
__global__ void bitonicSortStep(float* dev_values, int j, int k)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (dev_values[i] - dev_values[ixj] > 0) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i] - dev_values[ixj] < 0) {
        /* exchange(i,ixj); */
        float temp = dev_values[i];
        dev_values[i] = dev_values[ixj];
        dev_values[ixj] = temp;
      }
    }
  }
}

__global__ void fill(float* destination, size_t start, size_t destination_length, float fill_value){
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if(i + start >= destination_length){
    return;
  }

  destination[i + start] = fill_value;
}

/**
* Inplace bitonic sort using CUDA.
*/
void bitonicSortPow2(float* dev_values, size_t length)
{
  dim3 blocks(length > 64 ? 64 : 1, 1);
  dim3 threads(length > 64 ? length / 64 : length, 1);

  int j, k;
  /* Major step */
  for (k = 2; k <= length; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      bitonicSortStep<<<blocks, threads>>>(dev_values, j, k);
    }
  }
}

size_t nextPow2(size_t num){
  float log2 = log(num) / log(2);
  int exp = log2 == (int)log2 ? (int)log2 : (int)log2 + 1;
  return (size_t)pow(2, exp);
}

float* createBuffer(float* values, size_t values_length, size_t buffer_length, float fill_value){
  float* buffer = NULL;
  deviceMalloc((void**)&buffer, buffer_length * sizeof(float));
  deviceCopy(buffer, values, values_length * sizeof(float), hipMemcpyHostToDevice);

  size_t len_to_fill = buffer_length - values_length;
  if(len_to_fill > 0){
    dim3 blocks(len_to_fill > 64 ? 64 : 1, 1);
    dim3 threads(len_to_fill > 64 ? len_to_fill / 64 : len_to_fill, 1);
    fill<<<blocks, threads>>>(buffer, values_length, buffer_length, fill_value);
  }

  return buffer;
}

void bitonicSort(float* values, size_t length, float max_val){
  size_t valid_size = nextPow2(length);
  float* buffer = createBuffer(values, length, valid_size, max_val);

  bitonicSortPow2(buffer, valid_size);

  deviceCopy(values, buffer, length * sizeof(float), hipMemcpyDeviceToHost);
  deviceFree(buffer);
}

///
/// INT, INT
///
struct key_val_buffer{
  int* keys;
  int* values;
};

__global__ void bitonicSortPairsStep(int* keys, int* values, int j, int k, size_t length)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;

  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      if (keys[i] - keys[ixj] > 0) {
        /* exchange(i,ixj); */
        int temp_key = keys[i];
        int temp_val = values[i];
        keys[i] = keys[ixj];
        values[i] = values[ixj];
        keys[ixj] = temp_key;
        values[ixj] = temp_val;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (keys[i] - keys[ixj] < 0) {
        /* exchange(i,ixj); */
        int temp_key = keys[i];
        int temp_val = values[i];
        keys[i] = keys[ixj];
        values[i] = values[ixj];
        keys[ixj] = temp_key;
        values[ixj] = temp_val;
      }
    }
  }
}

__global__ void fillPairs(int* destination_a, int* destination_b, size_t start, size_t destination_length, int fill_a, int fill_b){
  int i = threadIdx.x + blockDim.x * blockIdx.x;

  if(i + start >= destination_length){
    return;
  }

  destination_a[i + start] = fill_a;
  destination_b[i + start] = fill_b;
}

void bitonicSortPairsPow2(int* keys, int* values, size_t length)
{
  kernelConfig kernel_size = calculateKernelConfig(length, MAX_THREADS_PER_BLOCK);

  int j, k;
  /* Major step */
  for (k = 2; k <= length; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      bitonicSortPairsStep<<<kernel_size.blocks, kernel_size.threads>>>(keys, values, j, k, length);
      deviceCheckErrors("bitonicSortPairsStep");
    }
  }
}

key_val_buffer createPairsBuffer(int* keys, int* values, size_t pairs_length, size_t buffer_length, int fill_key, int fill_value, bool src_is_device){
  key_val_buffer buffer;

  int* buf_keys = NULL;
  deviceMalloc((void**)&buf_keys, buffer_length * sizeof(int));
  deviceCopy(buf_keys, keys, pairs_length * sizeof(int), src_is_device ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice);

  int* buf_vals = NULL;
  deviceMalloc((void**)&buf_vals, buffer_length * sizeof(int));
  deviceCopy(buf_vals, values, pairs_length * sizeof(int), src_is_device ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice);

  size_t len_to_fill = buffer_length - pairs_length;
  if(len_to_fill > 0){
    dim3 blocks(len_to_fill > 64 ? 64 : 1, 1);
    dim3 threads(len_to_fill > 64 ? len_to_fill / 64 : len_to_fill, 1);
    fillPairs<<<blocks, threads>>>(buf_keys, buf_vals, pairs_length, buffer_length, fill_key, fill_value);
    deviceCheckErrors("fillPairs");
  }

  buffer.keys = buf_keys;
  buffer.values = buf_vals;

  return buffer;
}

void bitonicSortPairs(int* keys, int* values, size_t length, int max_key, bool src_is_device){
  size_t valid_size = nextPow2(length);
  T("sort: createPairsBuffer()");
  key_val_buffer buffer = createPairsBuffer(keys, values, length, valid_size, max_key, -1, src_is_device);

  T("bitonicSortPairsPow2()");
  bitonicSortPairsPow2(buffer.keys, buffer.values, valid_size);

  T("sort: deviceCopy()");
  deviceCopy(keys, buffer.keys, length * sizeof(int), src_is_device ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost);
  deviceCopy(values, buffer.values, length * sizeof(int), src_is_device ? hipMemcpyDeviceToDevice : hipMemcpyDeviceToHost);
  deviceFree(buffer.keys);
  deviceFree(buffer.values);
}

#undef T
