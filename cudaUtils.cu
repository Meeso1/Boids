#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#ifndef CUDAUTILS
#define CUDAUTILS

void deviceMalloc(void** pointer, size_t size){
	hipError_t error = hipMalloc(pointer, size);
	printf("	malloc (%p)\n", *pointer);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector (error code: %s)\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void deviceCopy(void* destination, const void* source, size_t size, hipMemcpyKind kind){
	hipError_t error = hipMemcpy(destination, source, size, kind);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to copy vector (error code: %s)\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void deviceFree(void* pointer){
	printf("	free   (%p", pointer);
	hipError_t err = hipFree(pointer);
	printf(")\n");
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector (error code: %s)\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void deviceCheckErrors(char* name){
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch %s (error code %s)!\n", name, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

#endif