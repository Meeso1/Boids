#include "bitonicSort.cu"

int main(){
	size_t length = 100;
	int* keys   = (int*)malloc(length*sizeof(int));
	int* values = (int*)malloc(length*sizeof(int));
	for(int i = 0; i < length; i++){
		keys[i] = i;
		values[i] = i;
	}

	int* d_keys = NULL;
	deviceMalloc((void**)&d_keys, length*sizeof(int));
	deviceCopy(d_keys, keys, length*sizeof(int), hipMemcpyHostToDevice);
	int* d_vals = NULL;
	deviceMalloc((void**)&d_vals, length*sizeof(int));
	deviceCopy(d_vals, values, length*sizeof(int), hipMemcpyHostToDevice);

	for(int i = 0; i < 20; i++){
		key_val_buffer buf1 = create_pairs_buffer(d_keys, d_vals, length, next_pow_2(length), -1, -1, true);
		deviceCopy(keys, d_keys, length*sizeof(int), hipMemcpyDeviceToHost);
		deviceCopy(values, d_vals, length*sizeof(int), hipMemcpyDeviceToHost);
		deviceFree(buf1.keys);
		deviceFree(buf1.values);
	}
	printf("DONE\n");

	bitonic_sort_pairs(d_keys, d_vals, length, length, true);
	bitonic_sort_pairs(d_keys, d_vals, length, length, true);
	bitonic_sort_pairs(d_keys, d_vals, length, length, true);
	printf("Done 2\n");

	bitonic_sort_pairs(keys, values, length, length, false);
	bitonic_sort_pairs(keys, values, length, length, false);
	bitonic_sort_pairs(keys, values, length, length, false);
	printf("Done 3\n");
	return 0;
}