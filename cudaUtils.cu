#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#ifndef CUDAUTILS
#define CUDAUTILS

struct kernelConfig{
	size_t blocks;
	size_t threads;
};

void deviceMalloc(void** pointer, size_t size){
	hipError_t error = hipMalloc(pointer, size);
	printf("	malloc (%p)\n", *pointer);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector (error code: %s)\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void deviceCopy(void* destination, const void* source, size_t size, hipMemcpyKind kind){
	hipError_t error = hipMemcpy(destination, source, size, kind);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to copy vector (error code: %s)\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void deviceFree(void* pointer){
	printf("	free   (%p", pointer);
	fflush(stdout);
	hipError_t err = hipFree(pointer);
	printf(")\n");
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector (error code: %s)\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void deviceCheckErrors(const char* name){
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch %s (error code: %s)\n", name, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

kernelConfig calculateKernelConfig(size_t total_num_of_threads, size_t max_threads_per_block_x){
	if(total_num_of_threads <= max_threads_per_block_x){
		return {1, total_num_of_threads};
	}

	double b = total_num_of_threads / ((double) max_threads_per_block_x);
	size_t num_of_blocks = (int)b == b ? (int)b : (int)b + 1;

	return {num_of_blocks, max_threads_per_block_x};
}

#endif