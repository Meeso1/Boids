#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#ifndef CUDAUTILS
#define CUDAUTILS

void deviceMalloc(void** pointer, size_t size){
	hipError_t error = hipMalloc(pointer, size);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void deviceCopy(void* destination, const void* source, size_t size, hipMemcpyKind kind){
	hipError_t error = hipMemcpy(destination, source, size, kind);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to copy vector (error code %s)!\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void deviceFree(void* pointer){
	hipError_t err = hipFree(pointer);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

#endif