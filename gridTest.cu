#define DEBUG_MSG 1
#include "grid.cu"
#include "testUtils.cu"

void manual_test(){
	int grid_size = 50;
	int cell_size = 5;
	size_t length = 34;
	double x[] = {0,  0,  1,  1,  2,  2,  3,  3, 5,  5,  7,  7, 11, 11, 13, 13, 17, 17, 19, 19, 23, 23, 29, 29, 31, 31, 37, 37, 41, 41, 43, 43, 47, 47};
	double y[] = {50, 37, 24, 11, 48, 35, 22, 9, 46, 33, 20, 7, 44, 31, 18, 5,  42, 29, 16, 3,  40, 27, 14, 1,  38, 25, 12, 49, 36, 23, 10, 47, 34, 21};

	printf("index from [2, 3]: %d\n", getCellIndexFrom2dIndexes(2, 3, getGridResolution(grid_size, cell_size)));
	printf("index of (49, 40): %d\n\n", getCellIndex(49, 40, grid_size, cell_size));

	double* d_x = NULL;
	double* d_y = NULL;
	deviceMalloc((void**)&d_x, length*sizeof(double)); 
	deviceMalloc((void**)&d_y, length*sizeof(double)); 
	deviceCopy(d_x, x, length*sizeof(double), hipMemcpyHostToDevice);
	deviceCopy(d_y, y, length*sizeof(double), hipMemcpyHostToDevice);

	printf("Creating grid...\n");
	Grid d_grid = makeGrid(grid_size, cell_size, length, d_x, d_y);
	printf("Grid created\n");
	Grid grid = copyToHost(d_grid);
	printf("Grid copied\n");

	printf("RAW GRID DATA:\n");
	printf("%zu indexes, %zu cells\n", grid.numOfIndexes, grid.numOfCells);
	printf("IDS:    ");
	print_int_array(grid.indexes.indexes, grid.numOfIndexes);
	printf("CELLS:  ");
	print_int_array(grid.indexes.cells, grid.numOfIndexes);
	printf("STARTS: ");
	print_int_array(grid.cellStarts, grid.numOfCells);

	printf("CELLS:\n");
	for(int cell = 0; cell < grid.numOfCells; cell++){
		int index = grid.cellStarts[cell];
		if(index == -1){
			// Cell is empty
			continue;
		}
		
		printf("{%3d}: ", cell);
		while(index < grid.numOfIndexes && grid.indexes.cells[index] == cell){
			int p = grid.indexes.indexes[index];
			printf("%3d(%4.1f, %4.1f) ", p, x[p], y[p]);
			index++;
		}
		printf("\n");
	}

	printf("NEIGHBOURS:\n");
	for(int i = 0; i < length; i++){
		int* neighbour_cells = getNeighbourCellsIndexesHost(getCellIndex(x[i], y[i], grid_size, cell_size), getGridResolution(grid_size, cell_size));
		printf("[%3d](%4.1f, %4.1f): ", i, x[i], y[i]);
		for(int k = 0; k < 9; k++){
			int cell = neighbour_cells[k];
			if(cell == -1) {
				continue; // No such cell
			}
			if(grid.cellStarts[cell] == -1){
				continue; // Cell is empty
			} 
			int index = grid.cellStarts[cell];
			while(index < grid.numOfIndexes && grid.indexes.cells[index] == cell){
				printf("%3d " , grid.indexes.indexes[index]);
				index++;
			}
		}
		printf("\n");
	}
}

void test_2d(){
	int grid_size = 100;
	int cell_size = 10;
	size_t length = 9;
	double x[] = {-1, 5, 5,  15, 15, 25, 45, 75, 85};
	double y[] = { 5, 5, 15, 5,  15, 25, 55, 85, 85};
	int res = 0;

	int resolution = getGridResolution(grid_size, cell_size);
	res += int_should_equal(resolution, 10);
	int test_index_1 = getCellIndexFrom2dIndexes(2, 3, resolution);
	res += int_should_equal(test_index_1, 32);
	int test_index_2 = getCellIndex(14, 41, grid_size, cell_size);
	res += int_should_equal(test_index_2, 41);

	double* d_x = NULL;
	double* d_y = NULL;
	deviceMalloc((void**)&d_x, length*sizeof(double)); 
	deviceMalloc((void**)&d_y, length*sizeof(double)); 
	deviceCopy(d_x, x, length*sizeof(double), hipMemcpyHostToDevice);
	deviceCopy(d_y, y, length*sizeof(double), hipMemcpyHostToDevice);

	Grid d_grid = makeGrid(grid_size, cell_size, length, d_x, d_y);
	Grid grid = copyToHost(d_grid);

	int ids_exp[] = {0, 1, 3, 2, 4, 5, 6, 7, 8};
	int cells_exp[] = {-1, 0, 1, 10, 11, 22, 54, 87, 88};

	int* starts_exp = (int*)malloc(100*sizeof(int));
	for(int i = 0; i < 100; i++){
		starts_exp[i] = -1;
	}
	for(int i = 0; i < length; i++){
		if(cells_exp[i] == -1) continue;
		if(starts_exp[cells_exp[i]] == -1) starts_exp[cells_exp[i]] = i;
	}

	res += int_should_equal(grid.numOfIndexes, length);
	res += int_should_equal(grid.numOfCells, 100);
	res += int_array_should_equal(grid.indexes.indexes, ids_exp, grid.numOfIndexes);
	res += int_array_should_equal(grid.indexes.cells, cells_exp, grid.numOfIndexes);
	res += int_array_should_equal(grid.cellStarts, starts_exp, grid.numOfCells);

	int index = grid.cellStarts[10];
	while(index < grid.numOfIndexes && grid.indexes.cells[index] == 10){
		index++;
	}
	res += int_should_equal(index, grid.cellStarts[10] + 1);

	int* neighbour_cells = getNeighbourCellsIndexesHost(getCellIndex(x[1], y[1], grid_size, cell_size), resolution);
	int count = 0;
	for(int k = 0; k < 9; k++){
		int cell = neighbour_cells[k];
		if(cell == -1) {
			continue; // No such cell
		}
		int index = grid.cellStarts[cell];
		if(index == -1){
			continue; // Cell is empty
		} 
		while(index < grid.numOfIndexes && grid.indexes.cells[index] == cell){
			count++;
			index++;
		}
	}
	res += int_should_equal(count, 4);
	freeGrid(d_grid);

	printf(res == 0 ? "PASSED\n" : "FAILED\n");
}

int main(){
	test_2d();
	return 0;
}