#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>

#ifndef CUDAUTILS
#define CUDAUTILS

size_t max_threads = 128;

struct kernelConfig{
	size_t blocks;
	size_t threads;
};

void deviceMalloc(void** pointer, size_t size){
	hipError_t error = hipMalloc(pointer, size);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to allocate device vector (error code: %s)\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void deviceCopy(void* destination, const void* source, size_t size, hipMemcpyKind kind){
	hipError_t error = hipMemcpy(destination, source, size, kind);
	if(error != hipSuccess){
		fprintf(stderr, "Failed to copy vector (error code: %s)\n", hipGetErrorString(error));
		exit(EXIT_FAILURE);
	}
}

void deviceFree(void* pointer){
	hipError_t err = hipFree(pointer);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to free device vector (error code: %s)\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void deviceCheckErrors(const char* name){
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to launch %s (error code: %s)\n", name, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void deviceSynchronize(){
	hipError_t err = hipDeviceSynchronize();
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to sync device (error code: %s)\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

kernelConfig calculateKernelConfig(size_t total_num_of_threads, size_t max_threads_per_block_x){
	if(total_num_of_threads <= max_threads_per_block_x){
		return {1, total_num_of_threads};
	}

	double b = total_num_of_threads / ((double) max_threads_per_block_x);
	size_t num_of_blocks = (int)b == b ? (int)b : (int)b + 1;

	return {num_of_blocks, max_threads_per_block_x};
}

void setMaxThreads(){
	int device;
	hipError_t err = hipGetDevice(&device);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to get device (error code: %s)\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	struct hipDeviceProp_t props;
	err = hipGetDeviceProperties(&props, device);
	if (err != hipSuccess) {
		fprintf(stderr, "Failed to get device properties (error code: %s)\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	max_threads = props.maxThreadsDim[0] / 2;
}

#endif